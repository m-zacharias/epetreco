#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>

#define M 6
#define N 4

#define IDX2C(i,j,ld) (((j)*(ld))+i)

int main()
{
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;

  float * A_host = 0;
  float * A_devi;
  float * x_host = 0;
  float * x_devi;
  float * y_host = 0;
  float * y_devi;
  
  /* Host memory alocation */
  A_host = (float *) malloc(M*N*sizeof(*A_host));
  if(!A_host) {
    std::cerr << "Host memory allocation failed" << std::endl;
    return EXIT_FAILURE;
  }
  for(int j=0; j<N; j++) {
    for(int i=0; i<M; i++) {
      A_host[IDX2C(i,j,M)] = (float)(i * M + j + 1);
    }
  }
  x_host = (float *) malloc(N*sizeof(*A_host));
  if(!A_host) {
    std::cerr << "Host memory allocation failed" << std::endl;
    return EXIT_FAILURE;
  }
  for(int j=0; j<N; j++) {
    x_host[j] = j;
  }
  y_host = (float *) malloc(M*sizeof(*A_host));
  if(!A_host) {
    std::cerr << "Host memory allocation failed" << std::endl;
    return EXIT_FAILURE;
  }
  for(int i=0; i<M; i++) {
    y_host[i] = 0;
  }
  
  
  
  /* Device memory allocation */
  cudaStat = hipMalloc((void**)&A_devi, M*N*sizeof(*A_host));
  if(cudaStat != hipSuccess) {
    std::err << "Device memory allocation failed" << std::endl;
    return EXIT_FAILURE;
  }
  cudaStat = hipMalloc((void**)&x_devi, N*sizeof(*A_host));
  if(cudaStat != hipSuccess) {
    std::err << "Device memory allocation failed" << std::endl;
    return EXIT_FAILURE;
  }
  cudaStat = hipMalloc((void**)&y_devi, M*sizeof(*A_host));
  if(cudaStat != hipSuccess) {
    std::err << "Device memory allocation failed" << std::endl;
    return EXIT_FAILURE;
  }
  
  
  
  /* CUBLAS initialization */
  stat = hipblasCreate(&handle);
  if(stat != CUBLASS_STATUS_SUCCESS) {
    std::err << "CUBLAS initialization failed" << std::endl;
    return EXIT_FAILURE;
  }
  
  
  
  /* Data download */
  stat = hipblasSetMatrix(M, N, sizeof(*A_host), A_host, M, A_devi, M);
  if(stat != CUBLASS_STATUS_SUCCESS) {
    std::err << "Data download failed" << std::endl;
    hipFree A_devi;
    hipFree x_devi;
    hipFree y_devi;
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
  stat = hipblasSetVector(N, sizeof(*A_host), x_host, 1, x_devi, 1);
  if(stat != CUBLASS_STATUS_SUCCESS) {
    std::err << "Data download failed" << std::endl;
    hipFree A_devi;
    hipFree x_devi;
    hipFree y_devi;
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
//  stat = hipblasSetVector(M, sizeof(*A_host), y_host, 1, y_devi, 1);
//  if(stat != CUBLASS_STATUS_SUCCESS) {
//    std::err << "Data download failed" << std::endl;
//    hipFree A_devi;
//    hipFree x_devi;
//    hipFree y_devi;
//    hipblasDestroy(handle);
//    return EXIT_FAILURE;
//  }
  
  
  
  /* Matrix Vector Multiplication */
  stat = hipblasSgemv(M, N, *alpha, A_devi, M, x_devi, 1, *beta, y_devi, 1);
  
  
  
  /* Data upload */
  stat = hipblasGetVector(M, sizeof(*A_host), y_devi, M, y_host, M);
  if(stat != CUBLASS_STATUS_SUCCESS) {
    std::err << "Data upload failed" << std::endl;
    hipFree A_devi;
    hipFree x_devi;
    hipFree y_devi;
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
  

  hipFree A_devi;
  hipFree x_devi;
  hipFree y_devi;
  hipblasDestroy(handle);
  for(int i=0; i<M; i++) {
    std::cout << y_host[IDX2C] << " ";
  }
  std::cout << std::endl;

  free(A_host);
  free(x_host);
  free(y_host);

  return EXIT_SUCCESS;
}
