#include "hip/hip_runtime.h"
/** @file   pureSMCalculation.cu */
/* Author: malte
 *
 * Created on 18. Februar 2015, 14:13 */

#define NBLOCKS 32

#include "wrappers.hpp"
#include "CUDA_HandleError.hpp"
#include "CUSPARSE_HandleError.hpp"
#include "csrmv.hpp"

/* [512 * 1024 * 1024 / 4] (512 MiB of float or int); max # of elems in COO
 * matrix arrays on GPU */
int const LIMNNZ(134217728);

/* Max # of channels in COO matrix arrays */
int const LIMM(LIMNNZ/VGRIDSIZE);

int main(int argc, char** argv) {
#ifdef MEASURE_TIME
  clock_t time1 = clock();
#endif
  int const nargs(2);
  if(argc!=nargs+1) {
    std::cerr << "Error: Wrong number of arguments. Exspected: "
              << nargs << ":" << std::endl
              << "  filename of measurement" << std::endl
              << "  number of rays" << std::endl;
    exit(EXIT_FAILURE);
  }
  std::string const fn(argv[1]);
  
  /* NUMBER OF RAYS PER CHANNEL */
  int const nrays(atoi(argv[2]));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(nrays_const), &nrays, sizeof(int)));
  
  /* MEASUREMENT SETUP */
  MS setup = MS(POS0X, POS1X, NA, N0Z, N0Y, N1Z, N1Y, DA, SEGX, SEGY, SEGZ);
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(setup_const), &setup, sizeof(MS)));
  
  /* VOXEL GRID */
  VG grid = VG(GRIDOX, GRIDOY, GRIDOZ, GRIDDX, GRIDDY, GRIDDZ, GRIDNX, GRIDNY, GRIDNZ);
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(grid_const), &grid, sizeof(grid)));
  
  /* MEASUREMENT LIST */
  /* Number of non-zeros, row indices */
  int effM; std::vector<int> yRowId_host;
  
  readMeasList_HDF5<val_t>(yRowId_host, effM, fn);
  
  int * yRowId_devi = NULL;
  HANDLE_ERROR(mallocMeasList_devi(yRowId_devi, effM));
  HANDLE_ERROR(cpyMeasListH2D(yRowId_devi, &(yRowId_host[0]), effM));
  
  /* MAX NUMBER OF NON_ZEROS IN SYSTEM MATRIX */
  int maxNnz = effM * VGRIDSIZE;
  
  /* SYSTEM MATRIX */
  /* Row (channel) ids, row pointers, effective row pointers, column (voxel)
   * ids, values, number of non-zeros (host, devi) */
  int * aCnlId_devi = NULL; int * aVxlId_devi = NULL; val_t * aVal_devi = NULL;
  HANDLE_ERROR(malloc_devi<int>(  aCnlId_devi, LIMM*VGRIDSIZE));
  HANDLE_ERROR(malloc_devi<int>(  aVxlId_devi, LIMM*VGRIDSIZE));
  HANDLE_ERROR(malloc_devi<val_t>(aVal_devi,   LIMM*VGRIDSIZE));
  int * nnz_devi = NULL;
  HANDLE_ERROR(malloc_devi<int>(  nnz_devi,    1));
#ifdef MEASURE_TIME
  clock_t time2 = clock();
  printTimeDiff(time2, time1, "Time before SM calculation: ");
#endif /* MEASURE_TIME */
  
  /* SM CALCULATION */
  for(int chunkId=0; chunkId<nChunks(maxNnz, LIMM*VGRIDSIZE); chunkId++) {
    int m = nInChunk(chunkId, effM, LIMM);
    int ptr = chunkPtr(chunkId, LIMM);
    
    int nnz_host[1] = {0};
    HANDLE_ERROR(memcpyH2D<int>(nnz_devi, nnz_host, 1));
    
    int * m_devi = NULL;
    HANDLE_ERROR(malloc_devi<int>(m_devi, 1));
    HANDLE_ERROR(memcpyH2D<int>(m_devi, &m, 1));
    
    /* Get system matrix */
    getSystemMatrix<
          val_t, VG, Idx, Idy, Idz, MS, Id0z, Id0y, Id1z, Id1y, Ida, Trafo0, Trafo1>
          <<<NBLOCKS, TPB>>>
          ( aVal_devi, aVxlId_devi, aCnlId_devi, &(yRowId_devi[ptr]), m_devi, nnz_devi);
    HANDLE_ERROR(hipDeviceSynchronize());
    
    /* Cleanup */
    hipFree(m_devi);
  }
#ifdef MEASURE_TIME
  clock_t time3 = clock();
  printTimeDiff(time3, time2, "Time for SM calculation: ");
#endif /* MEASURE_TIME */
          
  /* Cleanup */
  hipFree(yRowId_devi);
  hipFree(aCnlId_devi);
  hipFree(aVxlId_devi);
  hipFree(aVal_devi);
  hipFree(nnz_devi);
#ifdef MEASURE_TIME
  clock_t time4 = clock();
  printTimeDiff(time4, time3, "Time after SM calculation: ");
#endif /* MEASURE_TIME */
  
  return 0;
}

