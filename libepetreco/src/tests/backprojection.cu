#include "hip/hip_runtime.h"
/** @file backprojection.cu */
/* Author: malte
 *
 * Created on 16. Februar 2015, 11:28 */

#define NBLOCKS 32

#include "wrappers.hpp"
#include "CUDA_HandleError.hpp"
#include "CUSPARSE_HandleError.hpp"
#include "csrmv.hpp"
#include "mlemOperations.hpp"
#include "typedefs.hpp"

/* [512 * 1024 * 1024 / 4] (512 MiB of float or int); max # of elems in COO
 * matrix arrays on GPU */
MemArrSizeType const LIMNNZ(134217728);

/* Max # of channels in COO matrix arrays */
ListSizeType const LIMM(LIMNNZ/VGRIDSIZE);

int main(int argc, char** argv) {
#ifdef MEASURE_TIME
  clock_t time1 = clock();
#endif /* MEASURE_TIME */
  int const nargs(3);
  if(argc!=nargs+1) {
    std::cerr << "Error: Wrong number of arguments. Exspected: "
              << nargs << ":" << std::endl
              << "  filename of measurement" << std::endl
              << "  filename of output" << std::endl
              << "  number of rays" << std::endl;
    exit(EXIT_FAILURE);
  }
  std::string const fn(argv[1]);
  std::string const on(argv[2]);
  
  /* NUMBER OF RAYS PER CHANNEL */
  int const nrays(atoi(argv[3]));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(nrays_const), &nrays, sizeof(int)));
  
  /* MEASUREMENT SETUP */
  MS setup = MS(POS0X, POS1X, NA, N0Z, N0Y, N1Z, N1Y, DA, SEGX, SEGY, SEGZ);
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(setup_const), &setup, sizeof(MS)));
  
  /* VOXEL GRID */
  VG grid = VG(GRIDOX, GRIDOY, GRIDOZ, GRIDDX, GRIDDY, GRIDDZ, GRIDNX, GRIDNY, GRIDNZ);
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(grid_const), &grid, sizeof(grid)));
  
  /* MEASUREMENT VECTOR Y */
  /* Number of non-zeros, row indices, values. */
  ListSizeType effM; std::vector<int> yRowId_host; std::vector<val_t> yVal_host;
  
  do {
    int tmp_effM(0);
    readMeasVct_HDF5(yRowId_host, yVal_host, tmp_effM, fn);
    effM = ListSizeType(tmp_effM);
  } while(false);
  
  int * yRowId_devi = NULL;
  val_t * yVal_devi = NULL;
  HANDLE_ERROR(mallocSparseVct_devi(yRowId_devi, yVal_devi, effM));
  HANDLE_ERROR(cpySparseVctH2D(yRowId_devi, yVal_devi, &yRowId_host[0], &yVal_host[0], effM));

  
  /* STUFF FOR MV */
  hipsparseHandle_t handle = NULL; hipsparseMatDescr_t A = NULL;
  HANDLE_CUSPARSE_ERROR(hipsparseCreate(&handle));
  HANDLE_CUSPARSE_ERROR(hipsparseCreateMatDescr(&A));
  HANDLE_CUSPARSE_ERROR(customizeMatDescr(A, handle));
  val_t alpha = 1.; val_t beta = 1.;
  
  /* MAX NUMBER OF NON_ZEROS IN SYSTEM MATRIX */
  MemArrSizeType maxNnz(effM * VGRIDSIZE);
    
  /* DENSITY X */
  val_t x_host[VGRIDSIZE];
  for(int i=0; i<VGRIDSIZE; i++) { x_host[i] = 0.; }
  val_t * x_devi = NULL;
  HANDLE_ERROR(malloc_devi<val_t>(x_devi, VGRIDSIZE));
  HANDLE_ERROR(memcpyH2D<val_t>(x_devi, x_host, VGRIDSIZE));
  HANDLE_ERROR(hipDeviceSynchronize());
  
  /* SYSTEM MATRIX */
  /* Row (channel) ids, row pointers, effective row pointers, column (voxel)
   * ids, values, number of non-zeros (host, devi) */
  int * aCnlId_devi = NULL; int * aCsrCnlPtr_devi = NULL;
  int * aEcsrCnlPtr_devi = NULL; int * aVxlId_devi = NULL;
  val_t * aVal_devi = NULL;
  HANDLE_ERROR(mallocSystemMatrix_devi<val_t>(aCnlId_devi, aCsrCnlPtr_devi,
        aEcsrCnlPtr_devi, aVxlId_devi, aVal_devi, NCHANNELS, LIMM, VGRIDSIZE));
  MemArrSizeType * nnz_devi = NULL;
  HANDLE_ERROR(malloc_devi<MemArrSizeType>(nnz_devi,          1));
#ifdef MEASURE_TIME
  clock_t time2 = clock();
  printTimeDiff(time2, time1, "Time before BP: ");
#endif /* MEASURE_TIME */
  
  /* BACKPROJECT */
  for(ChunkGridSizeType chunkId=0;
        chunkId<nChunks<ChunkGridSizeType, MemArrSizeType>(maxNnz, MemArrSizeType(LIMM*VGRIDSIZE));
        chunkId++) {
    ListSizeType m   = nInChunk(chunkId, effM, LIMM);
    ListSizeType ptr = chunkPtr(chunkId, LIMM);
    
    MemArrSizeType nnz_host[1] = {0};
    HANDLE_ERROR(memcpyH2D<MemArrSizeType>(nnz_devi, nnz_host, 1));
    
    /* Get system matrix */
    systemMatrixCalculation<val_t> (
          aEcsrCnlPtr_devi, aVxlId_devi, aVal_devi,
          nnz_devi,
          aCnlId_devi, aCsrCnlPtr_devi,
          &(yRowId_devi[ptr]), &m,
          handle);
    HANDLE_ERROR(hipDeviceSynchronize());
    HANDLE_ERROR(memcpyD2H<MemArrSizeType>(nnz_host, nnz_devi, 1));
    HANDLE_ERROR(hipDeviceSynchronize());
  
    /* Backproject measurement on grid */
    CSRmv<val_t>()(handle, HIPSPARSE_OPERATION_TRANSPOSE,
          m, VGRIDSIZE, *nnz_host, &alpha, A, aVal_devi, aEcsrCnlPtr_devi, aVxlId_devi,
          &(yVal_devi[ptr]), &beta, x_devi);
    HANDLE_ERROR(hipDeviceSynchronize());
  }
#ifdef MEASURE_TIME
  clock_t time3 = clock();
  printTimeDiff(time3, time2, "Time for BP: ");
#endif /* MEASURE_TIME */
  
  /* Normalize */
  val_t norm = sum<val_t>(x_devi, VGRIDSIZE);
  std::cout << "Norm: " << norm << std::endl;
  HANDLE_ERROR(hipDeviceSynchronize());
  scales<val_t>(x_devi, val_t(1./norm), VGRIDSIZE);
  HANDLE_ERROR(hipDeviceSynchronize());
  
  /* Copy back to host */
  HANDLE_ERROR(memcpyD2H<val_t>(x_host, x_devi, VGRIDSIZE));
  HANDLE_ERROR(hipDeviceSynchronize());
  
  /* Write to file */
  writeDensity_HDF5(x_host, on, grid);
  
  /* Cleanup */
  hipFree(yRowId_devi);
  hipFree(yVal_devi);
  hipsparseDestroy(handle);
  hipsparseDestroyMatDescr(A);
  hipFree(x_devi);
  hipFree(aCnlId_devi);
  hipFree(aCsrCnlPtr_devi);
  hipFree(aEcsrCnlPtr_devi);
  hipFree(aVxlId_devi);
  hipFree(aVal_devi);
  hipFree(nnz_devi);
#ifdef MEASURE_TIME
  clock_t time4 = clock();
  printTimeDiff(time4, time3, "Time after BP: ");
#endif /* MEASURE_TIME */
  
  return 0;
}

