#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>

#define M 6 // #lines
#define N 4 // #cols

#define IDX2C(i,j,ld) (((j)*(ld))+i)

int main()
{
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;

  float * A_host = 0;
  float * A_devi;
  float * x_host = 0;
  float * x_devi;
  float * y_host = 0;
  float * y_devi;
  
  float alpha = 1.;
  float beta = 0.;

  /* Host memory alocation */
  A_host = (float *) malloc(M*N*sizeof(*A_host));
  if(!A_host) {
    std::cerr << "Host memory allocation failed" << std::endl;
    return EXIT_FAILURE;
  }
  for(int j=0; j<N; j++) {    /* iterate over cols */
    for(int i=0; i<M; i++) {  /* iterate over lines */
      A_host[IDX2C(i,j,M)] = i+1;
    }
  }
  x_host = (float *) malloc(N*sizeof(*A_host));
  if(!A_host) {
    std::cerr << "Host memory allocation failed" << std::endl;
    return EXIT_FAILURE;
  }
  for(int j=0; j<N; j++) {
    x_host[j] = 1;
  }
  y_host = (float *) malloc(M*sizeof(*A_host));
  if(!A_host) {
    std::cerr << "Host memory allocation failed" << std::endl;
    return EXIT_FAILURE;
  }
  for(int i=0; i<M; i++) {
    y_host[i] = 0;
  }
  
  
  
  /* Device memory allocation */
  cudaStat = hipMalloc((void**)&A_devi, M*N*sizeof(*A_host));
  if(cudaStat != hipSuccess) {
    std::cerr << "Device memory allocation failed" << std::endl;
    return EXIT_FAILURE;
  }
  cudaStat = hipMalloc((void**)&x_devi, N*sizeof(*A_host));
  if(cudaStat != hipSuccess) {
    std::cerr << "Device memory allocation failed" << std::endl;
    return EXIT_FAILURE;
  }
  cudaStat = hipMalloc((void**)&y_devi, M*sizeof(*A_host));
  if(cudaStat != hipSuccess) {
    std::cerr << "Device memory allocation failed" << std::endl;
    return EXIT_FAILURE;
  }
  
  
  
  /* CUBLAS initialization */
  stat = hipblasCreate(&handle);
  if(stat != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "CUBLAS initialization failed" << std::endl;
    return EXIT_FAILURE;
  }
  
  
  
  /* Data download */
  stat = hipblasSetMatrix(M, N, sizeof(*A_host), A_host, M, A_devi, M);
  if(stat != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "Matrix data download failed" << std::endl;
    hipFree(A_devi);
    hipFree(x_devi);
    hipFree(y_devi);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
  stat = hipblasSetVector(N, sizeof(*A_host), x_host, 1, x_devi, 1);
  if(stat != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "x Vector data download failed" << std::endl;
    hipFree(A_devi);
    hipFree(x_devi);
    hipFree(y_devi);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
//  stat = cublasSetVector(M, sizeof(*A_host), y_host, 1, y_devi, 1);
//  if(stat != CUBLAS_STATUS_SUCCESS) {
//    std::err << "Data download failed" << std::endl;
//    cudaFree(A_devi);
//    cudaFree(x_devi);
//    cudaFree(y_devi);
//    cublasDestroy(handle);
//    return EXIT_FAILURE;
//  }
  
  
  
  /* Matrix Vector Multiplication */
  stat = hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, A_devi, M, x_devi, 1, &beta, y_devi, 1);
  if(stat != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "Matrix Vector Multiplication failed" << std::endl;
    hipFree(A_devi);
    hipFree(x_devi);
    hipFree(y_devi);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
 
 
  
  /* Data upload */
  stat = hipblasGetVector(M, sizeof(*A_host), y_devi, 1, y_host, 1);
  if(stat != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "Data upload failed" << std::endl;
    hipFree(A_devi);
    hipFree(x_devi);
    hipFree(y_devi);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
  

  hipFree(A_devi);
  hipFree(x_devi);
  hipFree(y_devi);
  hipblasDestroy(handle);
  for(int i=0; i<M; i++) {
    std::cout << y_host[i] << " ";
  }
  std::cout << std::endl;

  free(A_host);
  free(x_host);
  free(y_host);

  return EXIT_SUCCESS;
}
