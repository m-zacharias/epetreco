/* Find out cuda shared memory
 */
#include <hip/hip_runtime.h>
#include "CUDA_HandleError.hpp"
#include <iostream>

int main()
{
  hipDeviceProp_t prop;
  HANDLE_ERROR( hipGetDeviceProperties( &prop, 0 ) );
  size_t mem = prop.sharedMemPerBlock;
  std::cout << "Maximum shared memory: "
            << mem / 1024 << "kB " << mem % 1024 << "B" << std::endl;

  return 0;
}
