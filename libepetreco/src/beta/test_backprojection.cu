#include "hip/hip_runtime.h"
/* Reconstruction program for real measurement data.  The reconstruction method
 * is Summed Backprojection.
 */

#include "real_measurementsetup_defines.h"
#include "voxelgrid52_defines.h"

#include "CUDA_HandleError.hpp"
#include "FileTalk.hpp"

#include "ChordsCalc_kernel3.cu"
#include "MeasurementSetup.hpp"
#include "VoxelGrid.hpp"
#include "CudaMS.hpp"
#include "CudaVG.hpp"
#include "CudaTransform.hpp"
#include "H5Reader.hpp"
#include "H5DensityWriter.hpp"
#include "visualization.hpp"
#include <iostream>
#include <iomanip>
#include <sstream>
#include <cstdlib>



template<typename T, typename ConcreteVoxelGrid>
class WriteableCudaVG : public CudaVG<T, ConcreteVoxelGrid>
{
  public:

    WriteableCudaVG(
          T const   gridO0, T const   gridO1, T const   gridO2,
          T const   gridD0, T const   gridD1, T const   gridD2,
          int const gridN0, int const gridN1, int const gridN2 )
    : CudaVG<T, ConcreteVoxelGrid>(
          gridO0, gridO1, gridO2,
          gridD0, gridD1, gridD2,
          gridN0, gridN1, gridN2) {}

    void getOrigin( float * origin )
    {
      for(int dim=0; dim<3; dim++)
        origin[dim] = this->hostRepr()->gridO[dim];
    }

    void getVoxelSize( float * voxelSize )
    {
      for(int dim=0; dim<3; dim++)
        voxelSize[dim] = this->hostRepr()->gridD[dim];
    }

    void getNumberOfVoxels( int * numberOfVoxels )
    {
      for(int dim=0; dim<3; dim++)
        numberOfVoxels[dim] = this->hostRepr()->gridN[dim];
    }
};



template<typename T>
struct MeasurementEvent
{
  T   _value;
  int _channel;
  
  __host__ __device__
  MeasurementEvent()
  : _value(0.), _channel(-1) {}

  __host__ __device__
  MeasurementEvent( T value_, int channel_)
  : _value(value_), _channel(channel_) {}

  __host__ __device__
  MeasurementEvent( MeasurementEvent<T> const & ori )
  {
    _value   = ori._value;
    _channel = ori._channel;
  }
  
  __host__ __device__
  ~MeasurementEvent()
  {}

  __host__ __device__
  void operator=( MeasurementEvent<T> const & rhs )
  {
    _value   = rhs._value;
    _channel = rhs._channel;
  }

  __host__ __device__
  T value() const
  {
    return _value;
  }

  __host__ __device__
  int channel() const
  {
    return _channel;
  }
};



//#define CHUNKSIZE 400000
//#define CHUNKSIZE 100                             // number of lines in one chunk
//#define RANDOM_SEED 1234
//#define NTHREADRAYS 100

typedef float val_t;

int main( int ac, char ** av )
{
  /* ---------------------------
   * Treat commandline arguments 
   * --------------------------- */
  SAYLINES(__LINE__-3, __LINE__-1);
  
  if(ac < 5)
  {
    std::cerr << "Wrong number of arguments. Exspected arguments:" << std::endl
              << "    1.: measurement filename (mandatory)" << std::endl
              << "    2.: chunkSize (mandatory)" << std::endl
              << "    3.: randomSeed (mandatory)" << std::endl
              << "    4.: nThreadRays (mandatory)" << std::endl
              << "    5.: file output prefix (optional, defaults to \"real_algo_output\")"
              << std::endl;
    exit(EXIT_FAILURE);
  }
  std::string fn(av[1]);
  int const   chunkSize(atoi(av[2]));
  int const   randomSeed(atoi(av[3]));
  int const   nThreadRays(atoi(av[4]));

  std::string outpre;
  if(ac >= 6)
    outpre = std::string(av[5]);
  else
    outpre = std::string("real_algo_output");
  

  /* --------------
   * Create objects
   * -------------- */
  SAYLINES(__LINE__-3, __LINE__-1);
  
  /* Voxel grid */
  WriteableCudaVG<val_t, DefaultVoxelGrid<val_t> > *
                            grid =

        new WriteableCudaVG<val_t, DefaultVoxelGrid<val_t> >(
              GRIDOX, GRIDOY, GRIDOZ,
              GRIDDX, GRIDDY, GRIDDZ,
              GRIDNX, GRIDNY, GRIDNZ);

  /* Measurement setup */
  CudaMS<val_t, DefaultMeasurementSetup<val_t> > *
                            setup =
        
        new CudaMS<val_t, DefaultMeasurementSetup<val_t> >(
              POS0X, POS1X,
              NA, N0Z, N0Y, N1Z, N1Y,
              DA, SEGX, SEGY, SEGZ);
  
  /* Transform (math object) */
  CudaTransform<val_t,val_t>
                            trafo;
 
  /* System matrix chunk */
  CudaMatrix<val_t,val_t>   chunk(chunkSize, VGRIDSIZE);
  
  for(int rowId=0; rowId<chunkSize; rowId++)
    for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
      chunk.set(rowId, vxlId, 0.);
  
  /* Measurement vector */
  CudaVector<val_t, val_t>  yValues_chunk(chunkSize);

  CudaVector<MeasurementEvent<val_t>, MeasurementEvent<val_t> > 
                            y_chunk(chunkSize); // chunk part of meas.

  for(int listId=0; listId<chunkSize; listId++)
    y_chunk.set(listId, MeasurementEvent<val_t>(0., -1));
  
  /* Density guess */
  CudaVector<val_t,val_t>   x(VGRIDSIZE);

  for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
    x.set(vxlId, 0.);
  
  /* Helper */
  val_t one(1.);
  val_t zero(0.);
 
  
  /* ----------------
   * Read measurement
   * ---------------- */
  SAYLINES(__LINE__-3, __LINE__-1);
  
  std::cout << "Total number of channels:" << std::endl
            << "    " << NCHANNELS << std::endl;
  
  /* Allocate memory for and read raw input data */
  H5Reader h5reader(fn);
  val_t * meas = new val_t[NCHANNELS];
  h5reader.read(meas);
  
  /* Count those channels, that have values != 0. */
  int count(0);
  for(int cnlId=0; cnlId<NCHANNELS; cnlId++)
    if(meas[cnlId] != 0.)
      count++;

  int const NEVENTS(count);
  int const NCHUNKS((NEVENTS+chunkSize-1)/chunkSize);
  
  std::cout << "Total number of events (non-zero channel values): " << std::endl
            << "    " << NEVENTS << std::endl;
  
  /* Create measurement vector */
  CudaVector<MeasurementEvent<val_t>, MeasurementEvent<val_t> >
                            y(NEVENTS);

  int listId(0);
  for(int cnlId=0; cnlId<NCHANNELS; cnlId++)
  {
    if(meas[cnlId] != 0.)
    {
       y.set(listId, MeasurementEvent<val_t>(meas[cnlId], cnlId));
       listId++;
    }
  }

#ifdef DEBUG  
  /* Print measurement vector */
  SAYLINE(__LINE__-1);
  std::cout << "y:"
            << std::endl;
  for(int listId=0; listId<NEVENTS; listId++)
  {
    MeasurementEvent<val_t> event = y.get(listId);
    std::stringstream ss("");
    ss << "listId " << listId << ": ("
       << event.channel() << ": " << event.value() << ")";
    std::cout << std::right
              << std::setw(15) << ss.str() << " "
              << std::endl;
  }
#endif  // DEBUG


  /* ----------------
   * Reconstruct
   * ---------------- */
  SAYLINES(__LINE__-3, __LINE__-1);

  /* Allocate memory for rays */
  val_t   rays_host[NCHUNKS*chunkSize*nThreadRays*6*sizeof(val_t)];
  val_t * rays_devi;
  HANDLE_ERROR( hipMalloc((void**)&rays_devi,
                NCHUNKS*chunkSize*nThreadRays*6*sizeof(val_t)) );
  
  /* Iterate over chunks */
  //for(int chunkId=0; (chunkId<UPPERCHUNKID) && (chunkId<NCHUNKS); chunkId++)
  for(int chunkId=0; chunkId<NCHUNKS; chunkId++)
  {
    /* Copy chunk's part of measurement vector */
    SAYLINE(__LINE__-1);
    
    for(int listId=0; listId<chunkSize; listId++)
    {
      MeasurementEvent<val_t> event;
      if(chunkId*chunkSize + listId < NEVENTS)
        event = y.get(chunkId*chunkSize + listId);
      else
        event = MeasurementEvent<val_t>(0., -1);

      assert(!isnan(event.value()));
      assert(!isinf(event.value()));
      //std::cout << "channel " << event.channel() << ": " << event.value()
      //          << std::endl;

      y_chunk.set(      listId, event);
      yValues_chunk.set(listId, event.value());
    }
    
#ifdef DEBUG 
    /* Print measurement vector */
    SAYLINE(__LINE__-1);
    std::cout << std::left
              << std::setw(16) << "y_chunk:"
              << std::setw(16) << "yValues_chunk:"
              << std::endl;
    for(int listId=0; listId<chunkSize; listId++)
    {
      MeasurementEvent<val_t> event = y_chunk.get(listId);
      val_t                   elem  = yValues_chunk.get(listId);
      std::stringstream ss("");
      ss << "(" << event.channel() << ": " << event.value() << ")";
      std::cout << std::right
                << std::setw(15) << ss.str() << " "
                << std::setw(15) << elem     << " "
                << std::endl;
    }
#endif  // DEBUG
    
    /* Set system matrix chunk's elements to null */
    SAYLINE(__LINE__-1);
    for(int listId=0; listId<chunkSize; listId++)
      for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
        chunk.set(listId, vxlId, 0.);
    HANDLE_ERROR( hipDeviceSynchronize() );

    /* Calculate system matrix chunk */
    SAYLINE(__LINE__-1);
    //chordsCalc_noVis(
    //      chunkId, NCHANNELS, chunkSize, 1,
    //      static_cast<val_t*>(chunk.data()),
    //      &y_chunk,
    //      grid,
    //      VGRIDSIZE,
    //      setup);
    chordsCalc(
          chunkId, NCHANNELS, chunkSize, 1,
          static_cast<val_t*>(chunk.data()),
          rays_devi,
          &y_chunk,
          grid,
          VGRIDSIZE,
          setup,
          randomSeed,
          nThreadRays);
    HANDLE_ERROR( hipDeviceSynchronize() );
    chunk.set_devi_data_changed();
    
    for(int listId=0; listId<chunkSize; listId++)
    {
      for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
      {
        val_t elem = chunk.get(listId, vxlId);
        assert(!isnan(elem));
        assert(!isinf(elem));
      }
    }

#ifdef DEBUG
    /* Print system matrix chunk */
    SAYLINE(__LINE__-1);
    std::cout << "chunk:" << std::endl;
    for(int listId=0; listId<chunkSize; listId++)
    {
      int count(0);
      for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
        if(chunk.get(listId, vxlId) != 0.) count++;

      if(count > 0)
      {
        std::cout << "  listId " << listId << ":  ";
        for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
        {
          val_t elem = chunk.get(listId, vxlId);
          if(elem != 0.)
            std::cout << elem << "  ";
        }
        std::cout << std::endl;
      }
    }
#endif  // DEBUG

    /* Back projection */
    SAYLINE(__LINE__-1);
    trafo.gemv(
          BLAS_OP_T,
          &one, &chunk,
          &yValues_chunk,
          &one, &x);
    x.set_devi_data_changed();

#ifdef DEBUG
    /* Print x */
    SAYLINE(__LINE__-1);
    std::cout << "x:" << std::endl;
    for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
      if(x.get(vxlId) != 0.)
        std::cout << "  " << x.get(vxlId);
    std::cout << std::endl;
#endif  // DEBUG
  } /* End iterate over chunks */


  /* ----------------
   * File output
   * ---------------- */
  SAYLINES(__LINE__-3, __LINE__-1);
  
  /* Write last guess */
  SAYLINE(__LINE__-1);
  val_t * guess = new val_t[VGRIDSIZE];
  for(int memid=0; memid<VGRIDSIZE; memid++)
    guess[memid] = x.get(memid);

  H5DensityWriter<WriteableCudaVG<val_t, DefaultVoxelGrid<val_t> > >
        h5writer(outpre + std::string("_x.h5"));
  
  h5writer.write(guess, *grid);
  
  /* Visualize grid */
  std::cout << hipGetErrorString(hipDeviceSynchronize()) << std::endl;
  SAYLINE(__LINE__-1);
  DefaultVoxelGrid<val_t> * hostRepr = grid->hostRepr();
  SAYLINE(__LINE__-1);
  PlyGrid<TemplateVertex<val_t> > visGrid("",
                          TemplateVertex<val_t>(hostRepr->gridO[0],
                                                hostRepr->gridO[1],
                                                hostRepr->gridO[2]),
                          hostRepr->gridN[0]+1,
                          hostRepr->gridN[1]+1,
                          hostRepr->gridN[2]+1,
                          hostRepr->gridD[0],
                          hostRepr->gridD[1],
                          hostRepr->gridD[2]);
  PlyWriter writer(outpre + std::string("_grid.ply"));
  writer.write(visGrid);
  writer.close();

  /* Visualize det0 */
  int   det0N[] = {1, N0Y, N0Z};
  val_t det0C[] = {POS0X, 0, 0};
  val_t detD[]  = {SEGX, SEGY, SEGZ};
  BetaPlyGrid<val_t> det0(
        "", det0C, detD, det0N, BetaPlyGrid<val_t>::AT_CENTER);
  PlyWriter det0Writer(outpre + std::string("_det0.ply"));
  det0Writer.write(det0);
  det0Writer.close();

  /* Visualize det1 */
  int   det1N[] = {1, N1Y, N1Z};
  val_t det1C[] = {POS1X, 0, 0};
  BetaPlyGrid<val_t> det1("", det1C, detD, det1N, BetaPlyGrid<val_t>::AT_CENTER);
  PlyWriter det1Writer(outpre + std::string("_det1.ply"));
  det1Writer.write(det1);
  det1Writer.close();

  /* Visualize rays */
  HANDLE_ERROR( hipMemcpy(rays_host, rays_devi,
                           6*NCHUNKS*chunkSize*nThreadRays*sizeof(val_t),
                           hipMemcpyDeviceToHost) );
  for(int i=0; i<NEVENTS; i++)
  {
    BetaCompositePlyGeom compositeLines("");
    BetaPlyLine<val_t> lines[nThreadRays];
    for(int idRay=0; idRay<nThreadRays; idRay++)
    {
      lines[idRay] = BetaPlyLine<val_t>("", &rays_host[6*(i*nThreadRays + idRay)]);
      compositeLines.add(&lines[idRay]);
    }

    std::stringstream fn("");
    fn << outpre << "_rays-" << std::setw(3) << std::setfill('0') << i << ".ply";
    PlyWriter raysWriter(fn.str());
    raysWriter.write(compositeLines);
    raysWriter.close();
  }


  return 0;
}
