#include "hip/hip_runtime.h"
/* Reconstruction program for real measurement data.  The reconstruction method
 * is Summed Backprojection.
 */

#ifndef MEASUREMENTSETUP_DEFINES
#define MEASUREMENTSETUP_DEFINES

#define N0Z 13        // 1st detector's number of segments in z
#define N0Y 13        // 1st detector's number of segments in y
#define N1Z 13        // 2nd detector's number of segments in z
#define N1Y 13        // 2nd detector's number of segments in y
#define NA  180       // number of angular positions
#define DA  2.        // angular step
#define POS0X -0.200  // position of 1st detector's center in x [m]
#define POS1X  0.700  // position of 2nd detector's center in x [m]
#define SEGX 0.02     // x edge length of one detector segment [m]
#define SEGY 0.004    // y edge length of one detector segment [m]
#define SEGZ 0.004    // z edge length of one detector segment [m]
#define NCHANNELS NA*N0Z*N0Y*N1Z*N1Y

#endif  // #define MEASUREMENTSETUP_DEFINES



#ifndef VOXELGRID_DEFINES
#define VOXELGRID_DEFINES

#define GRIDNX 4      // x dimension of voxel grid
#define GRIDNY 4      // y dimension of voxel grid
#define GRIDNZ 4      // z dimension od voxel grid
#define GRIDOX -0.05  // x origin of voxel grid [m]
#define GRIDOY -0.05  // y origin of voxel grid [m]
#define GRIDOZ -0.05  // z origin of voxel grid [m]
#define GRIDDX  0.025 // x edge length of one voxel [m]
#define GRIDDY  0.025 // y edge length of one voxel [m]
#define GRIDDZ  0.025 // z edge length of one voxel [m]
#define VGRIDSIZE GRIDNX*GRIDNY*GRIDNZ

#endif  // #define VOXELGRID_DEFINES



#include "real_defines.h"



#include "CUDA_HandleError.hpp"
#include "FileTalk.hpp"

#include "ChordsCalc_kernel3.cu"
#include "MeasurementSetup.hpp"
#include "VoxelGrid.hpp"
#include "CudaMS.hpp"
#include "CudaVG.hpp"
#include "CudaTransform.hpp"
#include "H5Reader.hpp"
#include "H5DensityWriter.hpp"
#include "visualization.hpp"
#include <iostream>
#include <iomanip>
#include <sstream>



template<typename T, typename ConcreteVoxelGrid>
class WriteableCudaVG : public CudaVG<T, ConcreteVoxelGrid>
{
  public:

    WriteableCudaVG(
          T const   gridO0, T const   gridO1, T const   gridO2,
          T const   gridD0, T const   gridD1, T const   gridD2,
          int const gridN0, int const gridN1, int const gridN2 )
    : CudaVG<T, ConcreteVoxelGrid>(
          gridO0, gridO1, gridO2,
          gridD0, gridD1, gridD2,
          gridN0, gridN1, gridN2) {}

    void getOrigin( float * origin )
    {
      for(int dim=0; dim<3; dim++)
        origin[dim] = this->hostRepr()->gridO[dim];
    }

    void getVoxelSize( float * voxelSize )
    {
      for(int dim=0; dim<3; dim++)
        voxelSize[dim] = this->hostRepr()->gridD[dim];
    }

    void getNumberOfVoxels( int * numberOfVoxels )
    {
      for(int dim=0; dim<3; dim++)
        numberOfVoxels[dim] = this->hostRepr()->gridN[dim];
    }
};



template<typename T>
struct MeasurementEvent
{
  T   _value;
  int _channel;
  
  __host__ __device__
  MeasurementEvent()
  : _value(0.), _channel(-1) {}

  __host__ __device__
  MeasurementEvent( T value_, int channel_)
  : _value(value_), _channel(channel_) {}

  __host__ __device__
  MeasurementEvent( MeasurementEvent<T> const & ori )
  {
    _value   = ori._value;
    _channel = ori._channel;
  }
  
  __host__ __device__
  ~MeasurementEvent()
  {}

  __host__ __device__
  void operator=( MeasurementEvent<T> const & rhs )
  {
    _value   = rhs._value;
    _channel = rhs._channel;
  }

  __host__ __device__
  T value() const
  {
    return _value;
  }

  __host__ __device__
  int channel() const
  {
    return _channel;
  }
};



//#define CHUNKSIZE 400000
#define CHUNKSIZE 100                             // number of lines in one chunk
#define UPPERCHUNKID ((NCHANNELS+CHUNKSIZE-1)/CHUNKSIZE)
//#define UPPERCHUNKID 1
#define RANDOM_SEED 1234
#define NTHREADRAYS 100

typedef float val_t;

int main( int ac, char ** av )
{
  /* ---------------------------
   * Treat commandline arguments 
   * --------------------------- */
  SAYLINES(__LINE__-3, __LINE__-1);
  
  if(ac < 2)
  {
    std::cerr << "Wrong number of arguments. Exspected arguments:" << std::endl
              << "    1.: measurement filename (mandatory)" << std::endl
              << "    2.: file output prefix (optional, defaults to \"real_algo_output\")"
              << std::endl;
    exit(EXIT_FAILURE);
  }
  std::string fn(av[1]);
  std::string outpre;
  if(ac >= 3)
    outpre = std::string(av[2]);
  else
    outpre = std::string("test_angular_index_counting");
  

  /* --------------
   * Create objects
   * -------------- */
  SAYLINES(__LINE__-3, __LINE__-1);
  
  /* Voxel grid */
  WriteableCudaVG<val_t, DefaultVoxelGrid<val_t> > *
                            grid =

        new WriteableCudaVG<val_t, DefaultVoxelGrid<val_t> >(
              GRIDOX, GRIDOY, GRIDOZ,
              GRIDDX, GRIDDY, GRIDDZ,
              GRIDNX, GRIDNY, GRIDNZ);

  /* Measurement setup */
  CudaMS<val_t, DefaultMeasurementSetup<val_t> > *
                            setup =
        
        new CudaMS<val_t, DefaultMeasurementSetup<val_t> >(
              POS0X, POS1X,
              NA, N0Z, N0Y, N1Z, N1Y,
              DA, SEGX, SEGY, SEGZ);
  
  /* Transform (math object) */
  CudaTransform<val_t,val_t>
                            trafo;
 
  /* System matrix chunk */
  CudaMatrix<val_t,val_t>   chunk(CHUNKSIZE, VGRIDSIZE);
  
  for(int rowId=0; rowId<CHUNKSIZE; rowId++)
    for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
      chunk.set(rowId, vxlId, 0.);
  
  /* Measurement vector */
  //CudaVector<val_t, val_t>  yValues_chunk(CHUNKSIZE);

  CudaVector<MeasurementEvent<val_t>, MeasurementEvent<val_t> > 
                            y_chunk(CHUNKSIZE); // chunk part of meas.

  for(int listId=0; listId<CHUNKSIZE; listId++)
    y_chunk.set(listId, MeasurementEvent<val_t>(0., -1));
 
  
  /* ----------------
   * Read measurement
   * ---------------- */
  SAYLINES(__LINE__-3, __LINE__-1);
  
  std::cout << "Total number of channels:" << std::endl
            << "    " << NCHANNELS << std::endl;
  
  /* Allocate memory for and read raw input data */
  H5Reader h5reader(fn);
  val_t * meas = new val_t[NCHANNELS];
  h5reader.read(meas);
  
  /* Count those channels, that have values != 0. */
  int count(0);
  for(int cnlId=0; cnlId<NCHANNELS; cnlId++)
    if(meas[cnlId] != 0.)
      count++;

  int const NEVENTS(count);
  int const NCHUNKS((NEVENTS+CHUNKSIZE-1)/CHUNKSIZE);
  
  std::cout << "Total number of events (non-zero channel values): " << std::endl
            << "    " << NEVENTS << std::endl;
  
  /* Create measurement vector */
  CudaVector<MeasurementEvent<val_t>, MeasurementEvent<val_t> >
                            y(NEVENTS);

  int listId(0);
  for(int cnlId=0; cnlId<NCHANNELS; cnlId++)
  {
    if(meas[cnlId] != 0.)
    {
       y.set(listId, MeasurementEvent<val_t>(meas[cnlId], cnlId));
       listId++;
    }
  }

#ifdef DEBUG  
/**//* Print measurement vector */
/**/SAYLINE(__LINE__-1);
/**/std::cout << "y:"
/**/          << std::endl;
/**/for(int listId=0; listId<NEVENTS; listId++)
/**/{
/**/  MeasurementEvent<val_t> event = y.get(listId);
/**/  std::stringstream ss("");
/**/  ss << "listId " << listId << ": ("
/**/     << event.channel() << ": " << event.value() << ")";
/**/  std::cout << std::right
/**/            << std::setw(15) << ss.str() << " "
/**/            << std::endl;
/**/}
#endif  // DEBUG


  /* -----------------------
   * Calculate system matrix
   * ----------------------- */
  SAYLINES(__LINE__-3, __LINE__-1);

  /* Allocate memory for rays */
  val_t   rays_host[NCHUNKS*CHUNKSIZE*NTHREADRAYS*6*sizeof(val_t)];
  val_t * rays_devi;
  HANDLE_ERROR( hipMalloc((void**)&rays_devi,
                NCHUNKS*CHUNKSIZE*NTHREADRAYS*6*sizeof(val_t)) );
  
  /* Iterate over chunks */
  for(int chunkId=0; (chunkId<UPPERCHUNKID) && (chunkId<NCHUNKS); chunkId++)
  {
    /* Copy chunk's part of measurement vector */
    SAYLINE(__LINE__-1);
    
    for(int listId=0; listId<CHUNKSIZE; listId++)
    {
      MeasurementEvent<val_t> event;
      if(chunkId*CHUNKSIZE + listId < NEVENTS)
        event = y.get(chunkId*CHUNKSIZE + listId);
      else
        event = MeasurementEvent<val_t>(0., -1);

      assert(!isnan(event.value()));
      assert(!isinf(event.value()));
      //std::cout << "channel " << event.channel() << ": " << event.value()
      //          << std::endl;

      y_chunk.set(      listId, event);
      //yValues_chunk.set(listId, event.value());
    }
    
#ifdef DEBUG 
/**//* Print measurement vector */
/**/SAYLINE(__LINE__-1);
/**/std::cout << std::left
/**/          << std::setw(16) << "y_chunk:"
/**/          << std::setw(16) << "yValues_chunk:"
/**/          << std::endl;
/**/for(int listId=0; listId<CHUNKSIZE; listId++)
/**/{
/**/  MeasurementEvent<val_t> event = y_chunk.get(listId);
/**/  val_t                   elem  = yValues_chunk.get(listId);
/**/  std::stringstream ss("");
/**/  ss << "(" << event.channel() << ": " << event.value() << ")";
/**/  std::cout << std::right
/**/            << std::setw(15) << ss.str() << " "
/**/            << std::setw(15) << elem     << " "
/**/            << std::endl;
/**/}
#endif  // DEBUG
    
    /* Set system matrix chunk's elements to null */
    SAYLINE(__LINE__-1);
    for(int listId=0; listId<CHUNKSIZE; listId++)
      for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
        chunk.set(listId, vxlId, 0.);
    HANDLE_ERROR( hipDeviceSynchronize() );

    /* Calculate system matrix chunk */
    SAYLINE(__LINE__-1);
    chordsCalc(
          chunkId, NCHANNELS, CHUNKSIZE, 1,
          static_cast<val_t*>(chunk.data()),
          rays_devi,
          &y_chunk,
          grid,
          VGRIDSIZE,
          setup,
          RANDOM_SEED,
          NTHREADRAYS);
    HANDLE_ERROR( hipDeviceSynchronize() );
    chunk.set_devi_data_changed();
    
    for(int listId=0; listId<CHUNKSIZE; listId++)
    {
      for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
      {
        val_t elem = chunk.get(listId, vxlId);
        assert(!isnan(elem));
        assert(!isinf(elem));
      }
    }

#ifdef DEBUG
/**//* Print system matrix chunk */
/**/SAYLINE(__LINE__-1);
/**/std::cout << "chunk:" << std::endl;
/**/for(int listId=0; listId<CHUNKSIZE; listId++)
/**/{
/**/  int count(0);
/**/  for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
/**/    if(chunk.get(listId, vxlId) != 0.) count++;
/**/
/**/  if(count > 0)
/**/  {
/**/    std::cout << "  listId " << listId << ":  ";
/**/    for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
/**/    {
/**/      val_t elem = chunk.get(listId, vxlId);
/**/      if(elem != 0.)
/**/        std::cout << elem << "  ";
/**/    }
/**/    std::cout << std::endl;
/**/  }
/**/}
#endif  // DEBUG
  } /* End iterate over chunks */


  /* ----------------
   * File output
   * ---------------- */
  SAYLINES(__LINE__-3, __LINE__-1);
  
  /* Visualize grid */
  std::cout << hipGetErrorString(hipDeviceSynchronize()) << std::endl;
  SAYLINE(__LINE__-1);
  DefaultVoxelGrid<val_t> * hostRepr = grid->hostRepr();
  SAYLINE(__LINE__-1);
  PlyGrid<TemplateVertex<val_t> > visGrid("",
                          TemplateVertex<val_t>(hostRepr->gridO[0],
                                                hostRepr->gridO[1],
                                                hostRepr->gridO[2]),
                          hostRepr->gridN[0]+1,
                          hostRepr->gridN[1]+1,
                          hostRepr->gridN[2]+1,
                          hostRepr->gridD[0],
                          hostRepr->gridD[1],
                          hostRepr->gridD[2]);
  PlyWriter writer(outpre + std::string("_grid.ply"));
  writer.write(visGrid);
  writer.close();

  /* Visualize det0 */
  int   det0N[] = {1, N0Y, N0Z};
  val_t det0C[] = {POS0X, 0, 0};
  val_t detD[]  = {SEGX, SEGY, SEGZ};
  BetaPlyGrid<val_t> det0(
        "", det0C, detD, det0N, BetaPlyGrid<val_t>::AT_CENTER);
  PlyWriter det0Writer(outpre + std::string("_det0.ply"));
  det0Writer.write(det0);
  det0Writer.close();

  /* Visualize det1 */
  int   det1N[] = {1, N1Y, N1Z};
  val_t det1C[] = {POS1X, 0, 0};
  BetaPlyGrid<val_t> det1("", det1C, detD, det1N, BetaPlyGrid<val_t>::AT_CENTER);
  PlyWriter det1Writer(outpre + std::string("_det1.ply"));
  det1Writer.write(det1);
  det1Writer.close();

  /* Visualize rays */
  HANDLE_ERROR( hipMemcpy(rays_host, rays_devi,
                           6*NCHUNKS*CHUNKSIZE*NTHREADRAYS*sizeof(val_t),
                           hipMemcpyDeviceToHost) );
  HANDLE_ERROR( hipDeviceSynchronize() );

  for(int eventId=0; eventId<NEVENTS; eventId++)
  {
    BetaCompositePlyGeom compositeLines("");
    BetaPlyLine<val_t> lines[NEVENTS*NTHREADRAYS];
    for(int rayId=0; rayId<NTHREADRAYS; rayId++)
    {
      lines[rayId] = BetaPlyLine<val_t>("", &rays_host[6*(eventId*NTHREADRAYS + rayId)]);
      compositeLines.add(&lines[rayId]);
    }

    std::stringstream fn("");
    fn << outpre << "_rays-" << std::setw(3) << std::setfill('0') << eventId << ".ply";
    PlyWriter raysWriter(fn.str());
    raysWriter.write(compositeLines);
    raysWriter.close();
  }


  return 0;
}
