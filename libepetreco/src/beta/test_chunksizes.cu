#include "hip/hip_runtime.h"
/* Reconstruction program for real measurement data.  The reconstruction method
 * is Summed Backprojection.
 */

#ifndef MEASUREMENTSETUP_DEFINES
#define MEASUREMENTSETUP_DEFINES

#define N0Z 13        // 1st detector's number of segments in z
#define N0Y 13        // 1st detector's number of segments in y
#define N1Z 13        // 2nd detector's number of segments in z
#define N1Y 13        // 2nd detector's number of segments in y
#define NA  180       // number of angular positions
#define DA  2.        // angular step
#define POS0X -0.457  // position of 1st detector's center in x [m]
#define POS1X  0.457  // position of 2nd detector's center in x [m]
#define SEGX 0.02     // x edge length of one detector segment [m]
#define SEGY 0.004    // y edge length of one detector segment [m]
#define SEGZ 0.004    // z edge length of one detector segment [m]
#define NCHANNELS NA*N0Z*N0Y*N1Z*N1Y

#endif  // #define MEASUREMENTSETUP_DEFINES



//#ifndef VOXELGRID_DEFINES
//#define VOXELGRID_DEFINES
//
//#define GRIDNX 4      // x dimension of voxel grid
//#define GRIDNY 4      // y dimension of voxel grid
//#define GRIDNZ 4      // z dimension od voxel grid
//#define GRIDOX -0.05  // x origin of voxel grid [m]
//#define GRIDOY -0.05  // y origin of voxel grid [m]
//#define GRIDOZ -0.05  // z origin of voxel grid [m]
//#define GRIDDX  0.025 // x edge length of one voxel [m]
//#define GRIDDY  0.025 // y edge length of one voxel [m]
//#define GRIDDZ  0.025 // z edge length of one voxel [m]
//#define VGRIDSIZE GRIDNX*GRIDNY*GRIDNZ
//
//#endif  // #define VOXELGRID_DEFINES
//---
//#ifndef VOXELGRID_DEFINES
//#define VOXELGRID_DEFINES
//
//#define GRIDNX 32       // x dimension of voxel grid
//#define GRIDNY 32       // y dimension of voxel grid
//#define GRIDNZ 32       // z dimension od voxel grid
//#define GRIDOX -0.10    // x origin of voxel grid [m]
//#define GRIDOY -0.10    // y origin of voxel grid [m]
//#define GRIDOZ -0.10    // z origin of voxel grid [m]
//#define GRIDDX  0.00625 // x edge length of one voxel [m]
//#define GRIDDY  0.00625 // y edge length of one voxel [m]
//#define GRIDDZ  0.00625 // z edge length of one voxel [m]
//#define VGRIDSIZE GRIDNX*GRIDNY*GRIDNZ
//
//#endif  // #define VOXELGRID_DEFINES
//---
#ifndef VOXELGRID_DEFINES
#define VOXELGRID_DEFINES

#define GRIDNX 52     // x dimension of voxel grid
#define GRIDNY 52     // y dimension of voxel grid
#define GRIDNZ 52     // z dimension od voxel grid
#define GRIDOX -0.026 // x origin of voxel grid [m]
#define GRIDOY -0.026 // y origin of voxel grid [m]
#define GRIDOZ -0.026 // z origin of voxel grid [m]
#define GRIDDX  0.001 // x edge length of one voxel [m]
#define GRIDDY  0.001 // y edge length of one voxel [m]
#define GRIDDZ  0.001 // z edge length of one voxel [m]
#define VGRIDSIZE GRIDNX*GRIDNY*GRIDNZ

#endif  // #define VOXELGRID_DEFINES



#include "real_defines.h"



#include "CUDA_HandleError.hpp"
#include "FileTalk.hpp"

#include "ChordsCalc_kernel3.cu"
#include "MeasurementSetup.hpp"
#include "VoxelGrid.hpp"
#include "CudaMS.hpp"
#include "CudaVG.hpp"
#include "CudaTransform.hpp"
#include "H5Reader.hpp"
#include "H5DensityWriter.hpp"
#include "visualization.hpp"
#include <iostream>
#include <iomanip>
#include <sstream>
#include <cstdlib>



template<typename T, typename ConcreteVoxelGrid>
class WriteableCudaVG : public CudaVG<T, ConcreteVoxelGrid>
{
  public:

    WriteableCudaVG(
          T const   gridO0, T const   gridO1, T const   gridO2,
          T const   gridD0, T const   gridD1, T const   gridD2,
          int const gridN0, int const gridN1, int const gridN2 )
    : CudaVG<T, ConcreteVoxelGrid>(
          gridO0, gridO1, gridO2,
          gridD0, gridD1, gridD2,
          gridN0, gridN1, gridN2) {}

    void getOrigin( float * origin )
    {
      for(int dim=0; dim<3; dim++)
        origin[dim] = this->hostRepr()->gridO[dim];
    }

    void getVoxelSize( float * voxelSize )
    {
      for(int dim=0; dim<3; dim++)
        voxelSize[dim] = this->hostRepr()->gridD[dim];
    }

    void getNumberOfVoxels( int * numberOfVoxels )
    {
      for(int dim=0; dim<3; dim++)
        numberOfVoxels[dim] = this->hostRepr()->gridN[dim];
    }
};



template<typename T>
struct MeasurementEvent
{
  T   _value;
  int _channel;
  
  __host__ __device__
  MeasurementEvent()
  : _value(0.), _channel(-1) {}

  __host__ __device__
  MeasurementEvent( T value_, int channel_)
  : _value(value_), _channel(channel_) {}

  __host__ __device__
  MeasurementEvent( MeasurementEvent<T> const & ori )
  {
    _value   = ori._value;
    _channel = ori._channel;
  }
  
  __host__ __device__
  ~MeasurementEvent()
  {}

  __host__ __device__
  void operator=( MeasurementEvent<T> const & rhs )
  {
    _value   = rhs._value;
    _channel = rhs._channel;
  }

  __host__ __device__
  T value() const
  {
    return _value;
  }

  __host__ __device__
  int channel() const
  {
    return _channel;
  }
};



#define UPPERCHUNKID 1

typedef float val_t;

int main( int ac, char ** av )
{
  std::cout << VGRIDSIZE << std::endl;
  /* ---------------------------
   * Treat commandline arguments 
   * --------------------------- */
  SAYLINES(__LINE__-3, __LINE__-1);
  
  if(ac < 5)
  {
    std::cerr << "Wrong number of arguments. Exspected arguments:" << std::endl
              << "    1.: measurement filename (mandatory)" << std::endl
              << "    2.: chunkSize (mandatory)" << std::endl
              << "    3.: randomSeed (mandatory)" << std::endl
              << "    4.: nThreadRays (mandatory)" << std::endl
              << "    5.: file output prefix (optional, defaults to \"real_algo_output\")"
              << std::endl;
    exit(EXIT_FAILURE);
  }
  std::string fn(av[1]);
  int const   chunkSize(atoi(av[2]));
  int const   randomSeed(atoi(av[3]));
  int const   nThreadRays(atoi(av[4]));

  std::string outpre;
  if(ac >= 6)
    outpre = std::string(av[5]);
  else
    outpre = std::string("real_algo_output");
  

  /* --------------
   * Create objects
   * -------------- */
  SAYLINES(__LINE__-3, __LINE__-1);
  
  /* Voxel grid */
  WriteableCudaVG<val_t, DefaultVoxelGrid<val_t> > *
                            grid =

        new WriteableCudaVG<val_t, DefaultVoxelGrid<val_t> >(
              GRIDOX, GRIDOY, GRIDOZ,
              GRIDDX, GRIDDY, GRIDDZ,
              GRIDNX, GRIDNY, GRIDNZ);

  /* Measurement setup */
  CudaMS<val_t, DefaultMeasurementSetup<val_t> > *
                            setup =
        
        new CudaMS<val_t, DefaultMeasurementSetup<val_t> >(
              POS0X, POS1X,
              NA, N0Z, N0Y, N1Z, N1Y,
              DA, SEGX, SEGY, SEGZ);
  
  /* Transform (math object) */
  CudaTransform<val_t,val_t>
                            trafo;
 
  /* System matrix chunk */
#ifdef WITH_CUDAMATRIX
  CudaMatrix<val_t,val_t>   chunk(chunkSize, VGRIDSIZE);
  
  for(int rowId=0; rowId<chunkSize; rowId++)
    for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
      chunk.set(rowId, vxlId, 0.);
#else
  val_t * chunk_host = 0;
  chunk_host = (val_t*)malloc(chunkSize*VGRIDSIZE*sizeof(val_t));
  val_t * chunk_devi = 0;
  HANDLE_ERROR( hipMalloc((void**)&chunk_devi,
                           chunkSize*VGRIDSIZE*sizeof(val_t)) );

  for(int rowId=0; rowId<chunkSize; rowId++)
    for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
      chunk_host[vxlId*chunkSize+rowId]=0.;
  HANDLE_ERROR( hipMemcpy(chunk_devi, chunk_host,
                           chunkSize*VGRIDSIZE*sizeof(val_t),
                           hipMemcpyHostToDevice) );
#endif
  
  /* Measurement vector */
  CudaVector<val_t, val_t>  yValues_chunk(chunkSize);

  CudaVector<MeasurementEvent<val_t>, MeasurementEvent<val_t> > 
                            y_chunk(chunkSize); // chunk part of meas.

  for(int listId=0; listId<chunkSize; listId++)
    y_chunk.set(listId, MeasurementEvent<val_t>(0., -1));
  
  /* Density guess */
  CudaVector<val_t,val_t>   x(VGRIDSIZE);

  for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
    x.set(vxlId, 0.);
  
  /* Helper */
  val_t one(1.);
  val_t zero(0.);
 
  
  /* ----------------
   * Read measurement
   * ---------------- */
  SAYLINES(__LINE__-3, __LINE__-1);
  
  std::cout << "Total number of channels:" << std::endl
            << "    " << NCHANNELS << std::endl;
  
  /* Allocate memory for and read raw input data */
  H5Reader h5reader(fn);
  val_t * meas = new val_t[NCHANNELS];
  h5reader.read(meas);
  
  /* Count those channels, that have values != 0. */
  int count(0);
  for(int cnlId=0; cnlId<NCHANNELS; cnlId++)
    if(meas[cnlId] != 0.)
      count++;

  int const NEVENTS(count);
  int const NCHUNKS((NEVENTS+chunkSize-1)/chunkSize);
  
  std::cout << "Total number of events (non-zero channel values): " << std::endl
            << "    " << NEVENTS << std::endl;
  
  /* Create measurement vector */
  CudaVector<MeasurementEvent<val_t>, MeasurementEvent<val_t> >
                            y(NEVENTS);

  int listId(0);
  for(int cnlId=0; cnlId<NCHANNELS; cnlId++)
  {
    if(meas[cnlId] != 0.)
    {
       y.set(listId, MeasurementEvent<val_t>(meas[cnlId], cnlId));
       listId++;
    }
  }

#ifdef DEBUG  
/**//* Print measurement vector */
/**/SAYLINE(__LINE__-1);
/**/std::cout << "y:"
/**/          << std::endl;
/**/for(int listId=0; listId<NEVENTS; listId++)
/**/{
/**/  MeasurementEvent<val_t> event = y.get(listId);
/**/  std::stringstream ss("");
/**/  ss << "listId " << listId << ": ("
/**/     << event.channel() << ": " << event.value() << ")";
/**/  std::cout << std::right
/**/            << std::setw(15) << ss.str() << " "
/**/            << std::endl;
/**/}
#endif  // DEBUG


  /* ----------------
   * Reconstruct
   * ---------------- */
  SAYLINES(__LINE__-3, __LINE__-1);

  /* Iterate over chunks */
  SAYLINE(__LINE__-1);
  //for(int chunkId=0; (chunkId<UPPERCHUNKID) && (chunkId<NCHUNKS); chunkId++)
  for(int chunkId=0; chunkId<NCHUNKS; chunkId++)
  {
    /* Copy chunk's part of measurement vector */
    SAYLINE(__LINE__-1);
    
    for(int listId=0; listId<chunkSize; listId++)
    {
      MeasurementEvent<val_t> event;
      if(chunkId*chunkSize + listId < NEVENTS)
        event = y.get(chunkId*chunkSize + listId);
      else
        event = MeasurementEvent<val_t>(0., -1);

      assert(!isnan(event.value()));
      assert(!isinf(event.value()));

      y_chunk.set(      listId, event);
      yValues_chunk.set(listId, event.value());
    }
    
#ifdef DEBUG 
/**//* Print measurement vector */
/**/SAYLINE(__LINE__-1);
/**/std::cout << std::left
/**/          << std::setw(16) << "y_chunk:"
/**/          << std::setw(16) << "yValues_chunk:"
/**/          << std::endl;
/**/for(int listId=0; listId<chunkSize; listId++)
/**/{
/**/  MeasurementEvent<val_t> event = y_chunk.get(listId);
/**/  val_t                   elem  = yValues_chunk.get(listId);
/**/  std::stringstream ss("");
/**/  ss << "(" << event.channel() << ": " << event.value() << ")";
/**/  std::cout << std::right
/**/            << std::setw(15) << ss.str() << " "
/**/            << std::setw(15) << elem     << " "
/**/            << std::endl;
/**/}
#endif  // DEBUG
    
    /* Set system matrix chunk's elements to null */
    SAYLINE(__LINE__-1);
#ifdef WITH_CUDAMATRIX
    for(int listId=0; listId<chunkSize; listId++)
      for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
        chunk.set(listId, vxlId, 0.);
    HANDLE_ERROR( hipDeviceSynchronize() );
#else
    for(int listId=0; listId<chunkSize; listId++)
      for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
        chunk_host[vxlId*chunkSize+listId]=0.;
    HANDLE_ERROR( hipMemcpy(chunk_devi, chunk_host,
                             chunkSize*VGRIDSIZE*sizeof(val_t),
                             hipMemcpyHostToDevice) );
#endif

    /* Calculate system matrix chunk */
    SAYLINE(__LINE__-1);
    chordsCalc_noVis(
          chunkId, NCHANNELS, chunkSize, 1,
#ifdef WITH_CUDAMATRIX
          static_cast<val_t*>(chunk.data()),
#else
          chunk_devi,
#endif
          &y_chunk,
          grid,
          VGRIDSIZE,
          setup,
          randomSeed,
          nThreadRays);
    HANDLE_ERROR( hipDeviceSynchronize() );
#ifdef WITH_CUDAMATRIX
    chunk.set_devi_data_changed();
#endif

//#ifndef WITH_CUDAMATRIX    
//    HANDLE_ERROR( hipMemcpy(chunk_host, chunk_devi,
//                             chunkSize*VGRIDSIZE*sizeof(val_t),
//                             hipMemcpyDeviceToHost) );
//#endif
//    for(int listId=0; listId<chunkSize; listId++)
//    {
//      for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
//      {
//#ifdef WITH_CUDAMATRIX
//        val_t elem = chunk.get(listId, vxlId);
//#else
//        val_t elem = chunk_host[vxlId*chunkSize+listId];
//#endif
//        assert(!isnan(elem));
//        assert(!isinf(elem));
//      }
//    }

#ifdef DEBUG
/**//* Print system matrix chunk */
/**/SAYLINE(__LINE__-1);
/**/std::cout << "chunk:" << std::endl;
/**/for(int listId=0; listId<chunkSize; listId++)
/**/{
/**/  int count(0);
/**/  for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
/**/    //if(chunk.get(listId, vxlId) != 0.) count++;
/**/    if(chunk_host[vxlId*chunkSize+listId] != 0.) count++;
/**/
/**/  if(count > 0)
/**/  {
/**/    std::cout << "  listId " << listId << ":  ";
/**/    for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
/**/    {
/**/      //val_t elem = chunk.get(listId, vxlId);
/**/      val_t elem = chunk_host[vxlId*chunkSize+listId];
/**/      if(elem != 0.)
/**/        std::cout << elem << "  ";
/**/    }
/**/    std::cout << std::endl;
/**/  }
/**/}
#endif  // DEBUG

#ifdef WITH_CUDAMATRIX
    /* Back projection */
    SAYLINE(__LINE__-1);
    trafo.gemv(
          BLAS_OP_T,
          &one, &chunk,
          &yValues_chunk,
          &one, &x);
    x.set_devi_data_changed();
#endif

#ifdef DEBUG
/**//* Print x */
/**/SAYLINE(__LINE__-1);
/**/std::cout << "x:" << std::endl;
/**/for(int vxlId=0; vxlId<VGRIDSIZE; vxlId++)
/**/  if(x.get(vxlId) != 0.)
/**/    std::cout << "  " << x.get(vxlId);
/**/std::cout << std::endl;
#endif  // DEBUG
  } /* End iterate over chunks */


  /* ----------------
   * File output
   * ---------------- */
  SAYLINES(__LINE__-3, __LINE__-1);
  
  /* Write last guess */
  SAYLINE(__LINE__-1);
  val_t * guess = new val_t[VGRIDSIZE];
  for(int memid=0; memid<VGRIDSIZE; memid++)
    guess[memid] = x.get(memid);

  H5DensityWriter<WriteableCudaVG<val_t, DefaultVoxelGrid<val_t> > >
        h5writer(outpre + std::string("_x.h5"));
  
  h5writer.write(guess, *grid);
  
  /* Visualize grid */
  std::cout << hipGetErrorString(hipDeviceSynchronize()) << std::endl;
  SAYLINE(__LINE__-1);
  DefaultVoxelGrid<val_t> * hostRepr = grid->hostRepr();
  SAYLINE(__LINE__-1);
  PlyGrid<TemplateVertex<val_t> > visGrid("",
                          TemplateVertex<val_t>(hostRepr->gridO[0],
                                                hostRepr->gridO[1],
                                                hostRepr->gridO[2]),
                          hostRepr->gridN[0]+1,
                          hostRepr->gridN[1]+1,
                          hostRepr->gridN[2]+1,
                          hostRepr->gridD[0],
                          hostRepr->gridD[1],
                          hostRepr->gridD[2]);
  PlyWriter writer(outpre + std::string("_grid.ply"));
  writer.write(visGrid);
  writer.close();

  /* Visualize det0 */
  int   det0N[] = {1, N0Y, N0Z};
  val_t det0C[] = {POS0X, 0, 0};
  val_t detD[]  = {SEGX, SEGY, SEGZ};
  BetaPlyGrid<val_t> det0(
        "", det0C, detD, det0N, BetaPlyGrid<val_t>::AT_CENTER);
  PlyWriter det0Writer(outpre + std::string("_det0.ply"));
  det0Writer.write(det0);
  det0Writer.close();

  /* Visualize det1 */
  int   det1N[] = {1, N1Y, N1Z};
  val_t det1C[] = {POS1X, 0, 0};
  BetaPlyGrid<val_t> det1("", det1C, detD, det1N, BetaPlyGrid<val_t>::AT_CENTER);
  PlyWriter det1Writer(outpre + std::string("_det1.ply"));
  det1Writer.write(det1);
  det1Writer.close();



  return 0;
}
