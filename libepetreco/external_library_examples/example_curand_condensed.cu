
// 2014-03-28
// http://docs.nvidia.com/cuda/curand/device-api-overview.html#device-api-example
/*
 * This program uses the device CURAND API to calculate what 
 * proportion of pseudo-random ints have low bit set.
 * It then generates uniform results to calculate how many
 * are greater than .5.
 * It then generates  normal results to calculate how many 
 * are within one standard deviation of the mean.
 */
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void setup_kernel( hiprandState * state )
{
  int id = threadIdx.x + blockIdx.x * 64;
  /* Each thread gets same seed, a different sequence 
     number, no offset */
  hiprand_init(1234, id, 0, &state[id]);
}

__global__ void generate_kernel(
      hiprandState *state,
      int n, 
      unsigned int *result)
{
  int id = threadIdx.x + blockIdx.x * 64;
  int count = 0;
  unsigned int x;
  /* Copy state to local memory for efficiency */
  hiprandState localState = state[id];
  /* Generate pseudo-random unsigned ints */
  for(int i = 0; i < n; i++) {
    x = hiprand(&localState);
    /* Check if low bit set */
    if(x & 1) {
      count++;
    }
  }
  /* Copy state back to global memory */
  state[id] = localState;
  /* Store results */
  result[id] += count;
}

__global__ void generate_uniform_kernel(
      hiprandState *state,
      int n, 
      unsigned int *result)
{
  int id = threadIdx.x + blockIdx.x * 64;
  unsigned int count = 0;
  float x;
  /* Copy state to local memory for efficiency */
  hiprandState localState = state[id];
  /* Generate pseudo-random uniforms */
  for(int i = 0; i < n; i++) {
    x = hiprand_uniform(&localState);
    /* Check if > .5 */
    if(x > .5) {
      count++;
    }
  }
  /* Copy state back to global memory */
  state[id] = localState;
  /* Store results */
  result[id] += count;
}

__global__ void generate_normal_kernel(
      hiprandState *state,
      int n, 
      unsigned int *result)
{
  int id = threadIdx.x + blockIdx.x * 64;
  unsigned int count = 0;
  float2 x;
  /* Copy state to local memory for efficiency */
  hiprandState localState = state[id];
  /* Generate pseudo-random normals */
  for(int i = 0; i < n/2; i++) {
    x = hiprand_normal2(&localState);
    /* Check if within one standard deviaton */
    if((x.x > -1.0) && (x.x < 1.0)) {
      count++;
    }
    if((x.y > -1.0) && (x.y < 1.0)) {
      count++;
    }
  }
  /* Copy state back to global memory */
  state[id] = localState;
  /* Store results */
  result[id] += count;
}



int main(int argc, char *argv[])
{
  int i;
  unsigned int total;
  hiprandState *devStates;
  unsigned int *devResults, *hostResults;
  int sampleCount = 10000;
  bool doubleSupported = 0;
  int device;
  struct hipDeviceProp_t properties;  

  /* check for double precision support */
  CUDA_CALL(hipGetDevice(&device));
  CUDA_CALL(hipGetDeviceProperties(&properties,device));
  if ( properties.major >= 2 || (properties.major == 1 && properties.minor >= 3) ) {
    doubleSupported = 1;
  }

  /* Allocate space for results on host */
  hostResults = (unsigned int *)calloc(64 * 64, sizeof(int));

  /* Allocate space for results on device */
  CUDA_CALL(hipMalloc((void **)&devResults, 64 * 64 * 
        sizeof(unsigned int)));

  /* Set results to 0 */
  CUDA_CALL(hipMemset(devResults, 0, 64 * 64 * 
        sizeof(unsigned int)));

  /* Allocate space for prng states on device */
  CUDA_CALL(hipMalloc((void **)&devStates, 64 * 64 * 
        sizeof(hiprandState)));
  
  /* Setup prng states */
  setup_kernel<<<64, 64>>>(devStates);
  
  /* Generate and use pseudo-random  */
  generate_kernel<<<64, 64>>>(devStates, sampleCount, devResults);
  
  
  
  /* Copy device memory to host */
  CUDA_CALL(hipMemcpy(hostResults, devResults, 64 * 64 * 
    sizeof(unsigned int), hipMemcpyDeviceToHost));

  /* Show result */
  total = 0;
  for(i = 0; i < 64 * 64; i++) {
    total += hostResults[i];
  }
  printf("Fraction with low bit set was %10.13f\n", 
    (float)total / (64.0f * 64.0f * sampleCount * 50.0f));
    
  /* Set results to 0 */
  CUDA_CALL(hipMemset(devResults, 0, 64 * 64 * 
        sizeof(unsigned int)));

  /* Generate and use uniform pseudo-random  */
  generate_uniform_kernel<<<64, 64>>>(devStates, sampleCount, devResults);
  
  

  /* Copy device memory to host */
  CUDA_CALL(hipMemcpy(hostResults, devResults, 64 * 64 * 
    sizeof(unsigned int), hipMemcpyDeviceToHost));

  /* Show result */
  total = 0;
  for(i = 0; i < 64 * 64; i++) {
    total += hostResults[i];
  }
  printf("Fraction of uniforms > 0.5 was %10.13f\n", 
    (float)total / (64.0f * 64.0f * sampleCount * 50.0f));
  /* Set results to 0 */
  CUDA_CALL(hipMemset(devResults, 0, 64 * 64 * 
        sizeof(unsigned int)));

  /* Generate and use uniform pseudo-random  */
  generate_normal_kernel<<<64, 64>>>(devStates, sampleCount, devResults);
  
  

  /* Copy device memory to host */
  CUDA_CALL(hipMemcpy(hostResults, devResults, 64 * 64 * 
    sizeof(unsigned int), hipMemcpyDeviceToHost));

  /* Show result */
  total = 0;
  for(i = 0; i < 64 * 64; i++) {
    total += hostResults[i];
  }
  printf("Fraction of normals within 1 standard deviation was %10.13f\n", 
    (float)total / (64.0f * 64.0f * sampleCount * 50.0f));

  /* Cleanup */
  CUDA_CALL(hipFree(devStates));
   
  CUDA_CALL(hipFree(devResults));
  free(hostResults);
  printf("^^^^ kernel_example PASSED\n");
  return EXIT_SUCCESS;
}
